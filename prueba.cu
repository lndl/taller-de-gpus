
#include <stdio.h>
#include <stdlib.h>


// Declaración de función para ver recursos del device
void devicenfo(void);

int main(int argc, char *argv[]){
	devicenfo();
	return(0);
}


//  Sacar por pantalla información del *device*
void devicenfo(void){
  struct hipDeviceProp_t capabilities;

  hipGetDeviceProperties (&capabilities, 0);

  printf("->CUDA Platform & Capabilities\n");
  printf("Name: %s\n", capabilities.name);
  printf("totalGlobalMem: %.2f MB\n", capabilities.totalGlobalMem/1024.0f/1024.0f);
  printf("sharedMemPerBlock: %.2f KB\n", capabilities.sharedMemPerBlock/1024.0f);
  printf("regsPerBlock (32 bits): %d\n", capabilities.regsPerBlock);
  printf("warpSize: %d\n", capabilities.warpSize);
  printf("memPitch: %.2f KB\n", capabilities.memPitch/1024.0f);
  printf("maxThreadsPerBlock: %d\n", capabilities.maxThreadsPerBlock);
  printf("maxThreadsDim: %d x %d x %d\n", capabilities.maxThreadsDim[0], 
	 capabilities.maxThreadsDim[1], capabilities.maxThreadsDim[2]);
  printf("maxGridSize: %d x %d\n", capabilities.maxGridSize[0], 
	 capabilities.maxGridSize[1]);
  printf("totalConstMem: %.2f KB\n", capabilities.totalConstMem/1024.0f);
  printf("major.minor: %d.%d\n", capabilities.major, capabilities.minor);
  printf("clockRate: %.2f MHz\n", capabilities.clockRate/1024.0f);
  printf("textureAlignment: %d\n", capabilities.textureAlignment);
  printf("deviceOverlap: %d\n", capabilities.deviceOverlap);
  printf("multiProcessorCount: %d\n", capabilities.multiProcessorCount);
}
